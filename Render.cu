#include "hip/hip_runtime.h"
#include "Render.cuh"

#include "Geometry.cuh"

#include <hiprand/hiprand_kernel.h>
#include <glad/gl.h>
#include <span>

unsigned CalcNumBlocks(const unsigned size, const unsigned blockSize)
{
    const unsigned count = size / blockSize;
    return size % blockSize ? count + 1 : count;
}

struct Scene {
    static Scene Create(const std::span<const Sphere>& spheres, const std::span<const Triangle>& triangles)
    {
        Scene scene {};
        hipMalloc(&scene.m_spheres, spheres.size_bytes());
        hipMalloc(&scene.m_triangles, triangles.size_bytes());
        hipMemcpy(scene.m_spheres, spheres.data(), spheres.size_bytes(), hipMemcpyHostToDevice);
        hipMemcpy(scene.m_triangles, triangles.data(), triangles.size_bytes(), hipMemcpyHostToDevice);
        scene.m_numSpheres = spheres.size();
        scene.m_numTriangles = triangles.size_bytes();
        return scene;
    }

    static void Free(const Scene& scene)
    {
        hipFree(scene.m_spheres);
        hipFree(scene.m_triangles);
    }

    __device__ float3 Trace(const Ray& ray) const
    {
        float closest = INFINITY;
        float3 normal { 0.0f, 0.0f, 0.0f };

        for (int i = 0; i < m_numSpheres; i++) {
            const Sphere& sphere = m_spheres[i];

            const float t = sphere.Hit(ray);
            if (t < 0.0f || t >= closest) {
                continue;
            }

            closest = t;
            normal = Normalize(ray.At(t) - sphere.center);
        }

        for (int i = 0; i < m_numTriangles; i++) {
            const Triangle& triangle = m_triangles[i];

            const float t = triangle.Hit(ray);
            if (t < 0.0f || t >= closest) {
                continue;
            }

            closest = t;
            normal = triangle.normal;
        }

        if (closest == INFINITY) {
            return Lerp(
                { 1.0f, 1.0f, 1.0f },
                { 0.5f, 0.7f, 1.0f },
                ray.direction.y * 0.5f + 0.5f);
        }

        return normal * 0.5f + 0.5f;
    }

    Sphere* m_spheres = nullptr;
    Triangle* m_triangles = nullptr;
    uint32_t m_numSpheres = 0;
    uint32_t m_numTriangles = 0;
};

// ReSharper disable once CppPassValueParameterByConstReference
__global__ void Render(const int width, const int height, float4* pixels, hiprandState* randomStates, const int samples, const Scene scene)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {
        return;
    }

    const int threadId = x + y * width;
    hiprandState* randomState = randomStates + threadId;
    hiprand_init(0, threadId, 0, randomState);

    const float aspectRatio = static_cast<float>(width) / static_cast<float>(height);

    float3 pixel { 0.0f, 0.0f, 0.0f };
    for (int i = 0; i < samples; i++) {
        const float2 uv = {
            (static_cast<float>(x) + hiprand_uniform(randomState) - 0.5f) / static_cast<float>(width),
            (static_cast<float>(y) + hiprand_uniform(randomState) - 0.5f) / static_cast<float>(height)
        };

        constexpr float3 origin { 0.0f, 0.0f, 2.0f };

        constexpr float focalLength = 1.0f;

        const float3 direction {
            aspectRatio * (uv.x - 0.5f) * 2.0f,
            -(uv.y - 0.5f) * 2.0f,
            -focalLength
        };

        const Ray ray { origin, Normalize(direction) };

        pixel = pixel + scene.Trace(ray);
    }
    pixel = pixel / samples;

    pixels[threadId] = { pixel.x, pixel.y, pixel.z, 1.0f };
}

void RenderImage(const int width, const int height)
{
    float4* pixels = nullptr;
    hiprandState* randomStates = nullptr;
    hipMallocManaged(&pixels, sizeof(float4) * width * height);
    hipMallocManaged(&randomStates, sizeof(hiprandState) * width * height);

    const Sphere spheres[] {
        { { 0.0f, 0.0f, 0.0f }, 1.0f },
    };

    const Triangle triangles[] {
        { { -2.0f, -1.0f, 2.0f },
            { 2.0f, -1.0f, 2.0f },
            { -2.0f, -1.0f, -2.0f } },
        { { -2.0f, -1.0f, -2.0f },
            { 2.0f, -1.0f, 2.0f },
            { 2.0f, -1.0f, -2.0f } },
    };

    const Scene scene = Scene::Create(spheres, triangles);

    const dim3 DIM_BLOCK {
        32,
        32,
        1
    };
    const dim3 DIM_GRID {
        CalcNumBlocks(width, DIM_BLOCK.x),
        CalcNumBlocks(height, DIM_BLOCK.y),
        1
    };
    Render<<<DIM_GRID, DIM_BLOCK>>>(width, height, pixels, randomStates, 64, scene);

    hipDeviceSynchronize();

    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_FLOAT, pixels);

    hipFree(pixels);
    hipFree(randomStates);
}
