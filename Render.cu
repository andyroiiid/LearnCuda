#include "hip/hip_runtime.h"
#include "Render.cuh"

#include "Geometry.cuh"

#include <GLFW/glfw3.h>
#include <glad/gl.h>
#include <span>

unsigned CalcNumBlocks(const unsigned size, const unsigned blockSize)
{
    const unsigned count = size / blockSize;
    return size % blockSize ? count + 1 : count;
}

Framebuffer Framebuffer::Create(const int width, const int height)
{
    Framebuffer framebuffer {};
    framebuffer.width = width;
    framebuffer.height = height;
    hipMallocManaged(&framebuffer.pixels, sizeof(float3) * width * height);
    hipMallocManaged(&framebuffer.randomStates, sizeof(hiprandState) * width * height);
    return framebuffer;
}

void Framebuffer::Free(const Framebuffer& framebuffer)
{
    hipFree(framebuffer.pixels);
    hipFree(framebuffer.randomStates);
}

struct Scene {
    static Scene Create(const std::span<const Sphere>& spheres, const std::span<const Triangle>& triangles)
    {
        Scene scene {};
        hipMalloc(&scene.m_spheres, spheres.size_bytes());
        hipMalloc(&scene.m_triangles, triangles.size_bytes());
        hipMemcpy(scene.m_spheres, spheres.data(), spheres.size_bytes(), hipMemcpyHostToDevice);
        hipMemcpy(scene.m_triangles, triangles.data(), triangles.size_bytes(), hipMemcpyHostToDevice);
        scene.m_numSpheres = spheres.size();
        scene.m_numTriangles = triangles.size_bytes();
        return scene;
    }

    static void Free(const Scene& scene)
    {
        hipFree(scene.m_spheres);
        hipFree(scene.m_triangles);
    }

    struct HitResult {
        float t;
        float3 position;
        float3 normal;
    };

    __device__ HitResult Hit(const Ray& ray) const
    {
        HitResult hit {
            INFINITY,
            { 0.0f, 0.0f, 0.0f },
            { 0.0f, 0.0f, 0.0f }
        };

        constexpr float CLOSEST_HIT = 0.001f;

        for (int i = 0; i < m_numSpheres; i++) {
            const Sphere& sphere = m_spheres[i];

            const float t = sphere.Hit(ray);
            if (t < CLOSEST_HIT || t >= hit.t) {
                continue;
            }

            hit.t = t;
            hit.position = ray.At(t);
            hit.normal = Normalize(hit.position - sphere.center);
        }

        for (int i = 0; i < m_numTriangles; i++) {
            const Triangle& triangle = m_triangles[i];

            const float t = triangle.Hit(ray);
            if (t < CLOSEST_HIT || t >= hit.t) {
                continue;
            }

            hit.t = t;
            hit.position = ray.At(t);
            hit.normal = triangle.normal;
        }

        return hit;
    }

    __device__ float3 Trace(Ray ray, const int maxBounces, hiprandState* randomState) const
    {
        int bounces = 0;
        while (bounces < maxBounces) {
            const HitResult hit = Hit(ray);
            if (hit.t == INFINITY) {
                break;
            }
            ray = { hit.position, hit.normal + RandomOnSphere(randomState) };
            bounces++;
        }

        const float3 skybox = Lerp(
            { 1.0f, 1.0f, 1.0f },
            { 0.5f, 0.7f, 1.0f },
            ray.direction.y * 0.5f + 0.5f);
        return skybox * powf(0.5f, bounces);
    }

    Sphere* m_spheres = nullptr;
    Triangle* m_triangles = nullptr;
    uint32_t m_numSpheres = 0;
    uint32_t m_numTriangles = 0;
};

__global__ void Render(
    // ReSharper disable once CppPassValueParameterByConstReference
    const Framebuffer framebuffer,
    const int maxBounces,
    // ReSharper disable once CppPassValueParameterByConstReference
    const Scene scene)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= framebuffer.width || y >= framebuffer.height) {
        return;
    }

    const int threadId = x + y * framebuffer.width;
    hiprandState* randomState = framebuffer.randomStates + threadId;
    hiprand_init(framebuffer.frameCount, threadId, 0, randomState);

    const float aspectRatio = static_cast<float>(framebuffer.width) / static_cast<float>(framebuffer.height);

    const float2 uv = {
        (static_cast<float>(x) + hiprand_uniform(randomState) - 0.5f) / static_cast<float>(framebuffer.width),
        (static_cast<float>(y) + hiprand_uniform(randomState) - 0.5f) / static_cast<float>(framebuffer.height)
    };

    constexpr float3 origin { 0.0f, 0.0f, 2.0f };

    constexpr float focalLength = 1.0f;

    const float3 direction {
        aspectRatio * (uv.x - 0.5f) * 2.0f,
        -(uv.y - 0.5f) * 2.0f,
        -focalLength
    };

    const Ray ray { origin, Normalize(direction) };

    float3& pixel = framebuffer.pixels[threadId];
    const float3 sample = scene.Trace(ray, maxBounces, randomState);
    pixel = Lerp(pixel, sample, 1.0f / static_cast<float>(framebuffer.frameCount + 1));
}

double RenderImage(const Framebuffer& framebuffer)
{
    const Sphere spheres[] {
        { { 0.0f, 0.0f, 0.0f }, 1.0f },
    };

    const Triangle triangles[] {
        { { -20.0f, -1.0f, 20.0f },
            { 20.0f, -1.0f, 20.0f },
            { -20.0f, -1.0f, -20.0f } },
        { { -20.0f, -1.0f, -20.0f },
            { 20.0f, -1.0f, 20.0f },
            { 20.0f, -1.0f, -20.0f } },
    };

    const Scene scene = Scene::Create(spheres, triangles);

    const dim3 DIM_BLOCK {
        32,
        32,
        1
    };
    const dim3 DIM_GRID {
        CalcNumBlocks(framebuffer.width, DIM_BLOCK.x),
        CalcNumBlocks(framebuffer.height, DIM_BLOCK.y),
        1
    };
    const double prevTime = glfwGetTime();
    Render<<<DIM_GRID, DIM_BLOCK>>>(framebuffer, 8, scene);
    hipDeviceSynchronize();
    const double currTime = glfwGetTime();

    glTexImage2D(
        GL_TEXTURE_2D,
        0,
        GL_RGB32F,
        framebuffer.width,
        framebuffer.height,
        0,
        GL_RGB,
        GL_FLOAT,
        framebuffer.pixels);

    return currTime - prevTime;
}
