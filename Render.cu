#include "hip/hip_runtime.h"
#include "Render.cuh"

#include "Geometry.cuh"

#include <glad/gl.h>
#include <span>

unsigned CalcNumBlocks(const unsigned size, const unsigned blockSize)
{
    const unsigned count = size / blockSize;
    return size % blockSize ? count + 1 : count;
}

struct Scene {
    static Scene Create(const std::span<const Sphere>& spheres, const std::span<const Triangle>& triangles)
    {
        Scene scene {};
        hipMalloc(&scene.m_spheres, spheres.size_bytes());
        hipMalloc(&scene.m_triangles, triangles.size_bytes());
        hipMemcpy(scene.m_spheres, spheres.data(), spheres.size_bytes(), hipMemcpyHostToDevice);
        hipMemcpy(scene.m_triangles, triangles.data(), triangles.size_bytes(), hipMemcpyHostToDevice);
        scene.m_numSpheres = spheres.size();
        scene.m_numTriangles = triangles.size_bytes();
        return scene;
    }

    static void Free(const Scene& scene)
    {
        hipFree(scene.m_spheres);
        hipFree(scene.m_triangles);
    }

    struct HitResult {
        float t;
        float3 position;
        float3 normal;
    };

    __device__ HitResult Hit(const Ray& ray) const
    {
        HitResult hit {
            INFINITY,
            { 0.0f, 0.0f, 0.0f },
            { 0.0f, 0.0f, 0.0f }
        };

        constexpr float CLOSEST_HIT = 0.001f;

        for (int i = 0; i < m_numSpheres; i++) {
            const Sphere& sphere = m_spheres[i];

            const float t = sphere.Hit(ray);
            if (t < CLOSEST_HIT || t >= hit.t) {
                continue;
            }

            hit.t = t;
            hit.position = ray.At(t);
            hit.normal = Normalize(hit.position - sphere.center);
        }

        for (int i = 0; i < m_numTriangles; i++) {
            const Triangle& triangle = m_triangles[i];

            const float t = triangle.Hit(ray);
            if (t < CLOSEST_HIT || t >= hit.t) {
                continue;
            }

            hit.t = t;
            hit.position = ray.At(t);
            hit.normal = triangle.normal;
        }

        return hit;
    }

    __device__ float3 Trace(Ray ray, const int maxBounces, hiprandState* randomState) const
    {
        int bounces = 0;
        while (bounces < maxBounces) {
            const HitResult hit = Hit(ray);
            if (hit.t == INFINITY) {
                break;
            }
            ray = { hit.position, hit.normal + RandomOnSphere(randomState) };
            bounces++;
        }

        const float3 skybox = Lerp(
            { 1.0f, 1.0f, 1.0f },
            { 0.5f, 0.7f, 1.0f },
            ray.direction.y * 0.5f + 0.5f);
        return skybox * powf(0.5f, bounces);
    }

    Sphere* m_spheres = nullptr;
    Triangle* m_triangles = nullptr;
    uint32_t m_numSpheres = 0;
    uint32_t m_numTriangles = 0;
};

__global__ void Render(
    const int width,
    const int height,
    float4* pixels,
    hiprandState* randomStates,
    const int samples,
    const int maxBounces,
    // ReSharper disable once CppPassValueParameterByConstReference
    const Scene scene)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {
        return;
    }

    const int threadId = x + y * width;
    hiprandState* randomState = randomStates + threadId;
    hiprand_init(0, threadId, 0, randomState);

    const float aspectRatio = static_cast<float>(width) / static_cast<float>(height);

    float3 pixel { 0.0f, 0.0f, 0.0f };
    for (int i = 0; i < samples; i++) {
        const float2 uv = {
            (static_cast<float>(x) + hiprand_uniform(randomState) - 0.5f) / static_cast<float>(width),
            (static_cast<float>(y) + hiprand_uniform(randomState) - 0.5f) / static_cast<float>(height)
        };

        constexpr float3 origin { 0.0f, 0.0f, 2.0f };

        constexpr float focalLength = 1.0f;

        const float3 direction {
            aspectRatio * (uv.x - 0.5f) * 2.0f,
            -(uv.y - 0.5f) * 2.0f,
            -focalLength
        };

        const Ray ray { origin, Normalize(direction) };

        pixel = pixel + scene.Trace(ray, maxBounces, randomState);
    }
    pixel = pixel / samples;

    pixels[threadId] = { pixel.x, pixel.y, pixel.z, 1.0f };
}

void RenderImage(const int width, const int height)
{
    float4* pixels = nullptr;
    hiprandState* randomStates = nullptr;
    hipMallocManaged(&pixels, sizeof(float4) * width * height);
    hipMallocManaged(&randomStates, sizeof(hiprandState) * width * height);

    const Sphere spheres[] {
        { { 0.0f, 0.0f, 0.0f }, 1.0f },
    };

    const Triangle triangles[] {
        { { -20.0f, -1.0f, 20.0f },
            { 20.0f, -1.0f, 20.0f },
            { -20.0f, -1.0f, -20.0f } },
        { { -20.0f, -1.0f, -20.0f },
            { 20.0f, -1.0f, 20.0f },
            { 20.0f, -1.0f, -20.0f } },
    };

    const Scene scene = Scene::Create(spheres, triangles);

    const dim3 DIM_BLOCK {
        32,
        32,
        1
    };
    const dim3 DIM_GRID {
        CalcNumBlocks(width, DIM_BLOCK.x),
        CalcNumBlocks(height, DIM_BLOCK.y),
        1
    };
    Render<<<DIM_GRID, DIM_BLOCK>>>(width, height, pixels, randomStates, 128, 8, scene);

    hipDeviceSynchronize();

    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_FLOAT, pixels);

    hipFree(pixels);
    hipFree(randomStates);
}
