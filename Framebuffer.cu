#include "hip/hip_runtime.h"
#include "Framebuffer.cuh"

#include <hiprand/hiprand_kernel.h>

Framebuffer Framebuffer::Create(const int width, const int height)
{
    Framebuffer framebuffer {};
    framebuffer.width = width;
    framebuffer.height = height;
    hipMallocManaged(&framebuffer.pixels, sizeof(float3) * width * height);
    hipMallocManaged(&framebuffer.randomStates, sizeof(hiprandState) * width * height);
    return framebuffer;
}

void Framebuffer::Free(const Framebuffer& framebuffer)
{
    hipFree(framebuffer.pixels);
    hipFree(framebuffer.randomStates);
}
